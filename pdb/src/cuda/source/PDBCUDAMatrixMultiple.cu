#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include "PDBCUDAMatrixMultiple.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

__global__ void matrixMulGPU(float *in1data,
                             unsigned int in1NumRow,
                             unsigned int in1NumCol,
                             float *in2data,
                             unsigned int in2NumRow,
                             unsigned int in2NumCol,
                             float *outdata) {
  if (in1NumCol != in2NumRow) {
    return;
  }
  unsigned int I = in1NumRow;
  unsigned int J = in2NumCol;
  unsigned int K = in1NumCol;

  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < I && col < J) {
    for (int k = 0; k < K; ++k) {
      outdata[row * J + col] += in1data[row * K + k] * in2data[k * J + col];
    }
  }
}

void copyFromHostToDevice(float **targetDevice, float *sourceHost, unsigned int numRows, unsigned int numCols) {
  const unsigned int numElems = numRows * numCols;
  gpuErrchk(hipMalloc((void **) targetDevice, numElems * sizeof(float)));
  gpuErrchk(hipMemcpy(*targetDevice, sourceHost, numElems * sizeof(float), hipMemcpyHostToDevice));
}

void copyFromDeviceToHost(float *targetHost, float *sourceDevice, unsigned int numRows, unsigned int numCols) {
  const unsigned int numElems = numRows * numCols;
  gpuErrchk(hipMemcpy(targetHost, sourceDevice, numElems * sizeof(float), hipMemcpyDeviceToHost));
}

void launchKernel(float *in1data,
                  unsigned int in1NumRow,
                  unsigned int in1NumCol,
                  float *in2data,
                  unsigned int in2NumRow,
                  unsigned int in2NumCol,
                  float *outdataGPU) {
  dim3 threads_per_block(2, 2, 1);
  dim3 number_of_blocks((in1NumRow / threads_per_block.x) + 1, (in2NumCol / threads_per_block.y) + 1, 1);

  matrixMulGPU <<< number_of_blocks, threads_per_block >>> (in1data, in1NumRow, in1NumCol, in2data, in2NumRow, in2NumCol, outdataGPU);
}

void initGPUMemoryToZero(float **memdata, unsigned int numRows, unsigned int numCols) {
  const unsigned int numElems = numRows * numCols;
  gpuErrchk(hipMalloc((void **) memdata, numElems * sizeof(float)));
  gpuErrchk(hipMemset(*memdata, 0, numElems * sizeof(float)));
}

void printCudaVersion() {
  std::cout << "CUDA Compiled version: " << __CUDACC_VER__ << std::endl;
  int runtime_ver;
  hipRuntimeGetVersion(&runtime_ver);
  std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;
  int driver_ver;
  hipDriverGetVersion(&driver_ver);
  std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}

